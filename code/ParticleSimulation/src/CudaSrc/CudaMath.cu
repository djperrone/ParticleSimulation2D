#include "hip/hip_runtime.h"
#include "CudaMath.cuh"

#define GLM_FORCE_CUDA
#include <glm/glm.hpp>
#include <glm/gtc/type_ptr.hpp>
#include <time.h>

#include "Novaura/CudaGLInterop/hip/hip_runtime_api.h"


#define NUM_THREADS 256
#define NUM_PARTICLES 1000
//#define NUM_BLOCKS (NUM_PARTICLES * 16 + NUM_THREADS - 1) / NUM_THREADS)

namespace CudaMath {	


	void __global__ CudaMath::MakeTranslationMatrices_gpu(Matrix44f* matrices, common::particle_t* particles, size_t numParticles)
	{
		int tid = blockDim.x * blockIdx.x + threadIdx.x;
		if (tid >= numParticles) return;

		MAKE_TRANSLATION_xyz(matrices[tid], (float)particles[tid].x, (float)particles[tid].y, 0.0f);
	}

	void CudaMath::MakeTranslationMatrices_cpu(Matrix44f* matrices, common::particle_t* particles, size_t numParticles)
	{
		int num_blocks = (numParticles + NUM_THREADS - 1) / NUM_THREADS;

		MakeTranslationMatrices_gpu CUDA_KERNEL(num_blocks, NUM_THREADS)(matrices, particles, numParticles);
		hipError_t cudaerr = hipDeviceSynchronize();
		if (cudaerr != hipSuccess)
		{
			printf("translation kernel launch failed with error \"%s\".\n",
				hipGetErrorString(cudaerr));
			exit(-1);
		}
	}	

	void CudaMath::MatMul44_cpu(Matrix44f* A, Matrix44f* B, Matrix44f* C, int N)
	{		
		for (int i = 0; i < 4; i++)
		{
			for (int j = 0; j < 4; j++)
			{
				for (int k = 0; k < 4; k++)
				{
					C->rows[i].vec[j] += A->rows[k].vec[j] * B->rows[i].vec[k];
				}
			}
		}
	}
	__global__ void MatMul44Batch_gpu(Matrix44f* grid, Matrix44f* B, Matrix44f* C, int numParticles)
	{	
		int tid = blockDim.x * blockIdx.x + threadIdx.x;		
		
		const int localSize = NUM_THREADS / 16;
		__shared__ Matrix44f localGrid[localSize];		

		__shared__ Matrix44f localB;
		memcpy(&localB, B, sizeof(Matrix44f));

		if (tid >= NUM_PARTICLES * 16 || blockIdx.x >= NUM_PARTICLES) return;

		memcpy(localGrid, grid + blockIdx.x * localSize, sizeof(Matrix44f) * localSize);

		int i = tid / 16;
		int j = tid % 16;

		int row = j / 4;
		int col = j % 4;		
		
		float tmpSum = 0;
		int local_i = i % localSize;
		//printf("blkidx: %i, i: %i, local_i: %i, local_size: %i\n",blockIdx.x,i ,local_i, localSize);

		tmpSum += localGrid[local_i].rows[0].vec[col] * localB.rows[row].vec[0];
		tmpSum += localGrid[local_i].rows[1].vec[col] * localB.rows[row].vec[1];
		tmpSum += localGrid[local_i].rows[2].vec[col] * localB.rows[row].vec[2];
		tmpSum += localGrid[local_i].rows[3].vec[col] * localB.rows[row].vec[3];

	/*	tmpSum += grid[i].rows[0].vec[col] * localB.rows[row].vec[0];
		tmpSum += grid[i].rows[1].vec[col] * localB.rows[row].vec[1];
		tmpSum += grid[i].rows[2].vec[col] * localB.rows[row].vec[2];
		tmpSum += grid[i].rows[3].vec[col] * localB.rows[row].vec[3];*/
		
		//__syncthreads();
		C[i].mat[j] = tmpSum;		

		/*for (int k = 0; k < 4; k++)
		{
			tmpSum += grid[i].rows[k].vec[col] * B->rows[row].vec[k];
		}*/
	}

	__global__ void MatMul44Batch_gpu(Matrix44f* inGrid, Matrix44f* B, Matrix44f* outGrid, int* numParticles)
	{
		int tid = blockDim.x * blockIdx.x + threadIdx.x;

	/*	const int localSize = NUM_THREADS / 16;
		__shared__ Matrix44f localGrid[localSize];

		__shared__ Matrix44f localB;
		memcpy(&localB, B, sizeof(Matrix44f));*/

		if (tid >= *numParticles * 16 || blockIdx.x >= *numParticles) return;

		//memcpy(localGrid, inGrid + blockIdx.x * localSize, sizeof(Matrix44f) * localSize);

		int i = tid / 16;
		int j = tid % 16;

		int row = j / 4;
		int col = j % 4;

		float tmpSum = 0;
		//int local_i = i % localSize;
		//printf("blkidx: %i, i: %i, local_i: %i, local_size: %i\n",blockIdx.x,i ,local_i, localSize);

		//tmpSum += localGrid[local_i].rows[0].vec[col] * localB.rows[row].vec[0];
		//tmpSum += localGrid[local_i].rows[1].vec[col] * localB.rows[row].vec[1];
		//tmpSum += localGrid[local_i].rows[2].vec[col] * localB.rows[row].vec[2];
		//tmpSum += localGrid[local_i].rows[3].vec[col] * localB.rows[row].vec[3];

			tmpSum += inGrid[i].rows[0].vec[col] * B->rows[row].vec[0];
			tmpSum += inGrid[i].rows[1].vec[col] * B->rows[row].vec[1];
			tmpSum += inGrid[i].rows[2].vec[col] * B->rows[row].vec[2];
			tmpSum += inGrid[i].rows[3].vec[col] * B->rows[row].vec[3];

			//__syncthreads();
		outGrid[i].mat[j] = tmpSum;

		/*for (int k = 0; k < 4; k++)
		{
			tmpSum += grid[i].rows[k].vec[col] * B->rows[row].vec[k];
		}*/
	}

	void MatMul44Batch_cpu(Matrix44f* inGrid, Matrix44f* B, Matrix44f* outGrid, int numParticles)
	{
		int num_blocks = (numParticles * 16 + NUM_THREADS - 1) / NUM_THREADS;
		int* numParticles_d;
		hipMalloc((void**)&numParticles_d, sizeof(int));
		hipMemcpy(numParticles_d, &numParticles, sizeof(int), hipMemcpyHostToDevice);

		MatMul44Batch_gpu CUDA_KERNEL(num_blocks, NUM_THREADS) (inGrid, B,outGrid, numParticles_d);
		hipError_t cudaerr = hipDeviceSynchronize();
		if (cudaerr != hipSuccess)
		{
			printf("matmul44batch kernel launch failed with error \"%s\".\n",
				hipGetErrorString(cudaerr));
			exit(-1);
		}

		hipFree(numParticles_d);
	}

	
	void MatMul44BatchTest_cpu()
	{
		for (int num_tests = 0; num_tests < 50; num_tests++)
		{
			srand((unsigned int)time(NULL));

			Matrix44f A[NUM_PARTICLES], B, C[NUM_PARTICLES];
			Matrix44f* A_d, * B_d, * C_d;


			glm::mat4 ref_mats[NUM_PARTICLES];

			float scale = (float)rand() / (float)(RAND_MAX / 55.0f);
			MAKE_SCALE(B, scale);

			glm::mat4 scale_ref = glm::scale(glm::mat4(1.0f), glm::vec3(scale));			

			for (int i = 0; i < NUM_PARTICLES; i++)
			{
				float x = (float)rand() / (float)(RAND_MAX / 55.0f);
				float y = (float)rand() / (float)(RAND_MAX / 55.0f);
				float z = (float)rand() / (float)(RAND_MAX / 55.0f);

				Vector3f transVec{ x, y, z };
				MAKE_TRANSLATION(A[i], transVec);

				ref_mats[i] = glm::translate(glm::mat4(1.0f), glm::vec3(x, y, z));
			}
			printf("test {%i}", __LINE__);

			for (int i = 0; i < NUM_PARTICLES; i++)
				ZERO_FLAT_MATRIX(C[i]);

			for (int k = 0; k < NUM_PARTICLES; k++)
				for (int i = 0; i < 4; i++)
					for (int j = 0; j < 4; j++)
					{
						if (A[k].rows[i].vec[j] != ref_mats[k][i][j])
						{
							printf("initial matrices dont match\n");
							return;
						}
					}
			printf("success\n");
			hipError_t cudaerr = hipDeviceSynchronize();
			if (cudaerr != hipSuccess)
			{
				printf("1 kernel launch failed with error \"%s\".\n",
					hipGetErrorString(cudaerr));
				__debugbreak;
			}

			hipMalloc((void**)&B_d, sizeof(Matrix44f));
			hipMalloc((void**)&A_d, sizeof(Matrix44f) * NUM_PARTICLES);
			hipMalloc((void**)&C_d, sizeof(Matrix44f) * NUM_PARTICLES);

			cudaerr = hipDeviceSynchronize();
			if (cudaerr != hipSuccess)
			{
				printf("2 kernel launch failed with error \"%s\".\n",
					hipGetErrorString(cudaerr));
				__debugbreak;
			}

			hipMemcpy(B_d, &B, sizeof(Matrix44f), hipMemcpyHostToDevice);
			hipMemcpy(A_d, A, sizeof(Matrix44f) * NUM_PARTICLES, hipMemcpyHostToDevice);
			hipMemcpy(C_d, C, sizeof(Matrix44f) * NUM_PARTICLES, hipMemcpyHostToDevice);


			int num_blocks = (NUM_PARTICLES * 16 + NUM_THREADS - 1) / NUM_THREADS;

			MatMul44Batch_gpu CUDA_KERNEL(num_blocks, NUM_THREADS) (A_d, B_d, C_d, NUM_PARTICLES);
			cudaerr = hipDeviceSynchronize();
			if (cudaerr != hipSuccess)
			{
				printf("4 kernel launch failed with error \"%s\".\n",
					hipGetErrorString(cudaerr));
				exit(-1);
			}

			hipMemcpy(A, A_d, sizeof(Matrix44f) * NUM_PARTICLES, hipMemcpyDeviceToHost);
			hipMemcpy(C, C_d, sizeof(Matrix44f) * NUM_PARTICLES, hipMemcpyDeviceToHost);


			for (int k = 0; k < NUM_PARTICLES; k++)
				for (int i = 0; i < 4; i++)
					for (int j = 0; j < 4; j++)
					{
						//printf("%f, ", A[k].rows[i].vec[j]);
						if (A[k].rows[i].vec[j] != ref_mats[k][i][j])
						{
							printf("after matrices dont match\n");
							exit(-1);
						}
					}
			printf("success3\n");


			glm::mat4 results[NUM_PARTICLES];
			for (int i = 0; i < NUM_PARTICLES; i++)
			{
				results[i] = ref_mats[i] * scale_ref;
			}

			for (int k = 0; k < NUM_PARTICLES; k++)
				for (int i = 0; i < 4; i++)
					for (int j = 0; j < 4; j++)
					{
						//printf("%f, ", A[k].rows[i].vec[j]);
						if (C[k].rows[i].vec[j] != results[k][i][j])
						{
							printf("after matrices dont match\n");
							exit(-1);
						}

					}
			printf("success!!!!!!\n");
		}		
	}

	__global__ void MatMul44_gpu(Matrix44f* A, Matrix44f* B, Matrix44f* C, int N)
	{
		int test = threadIdx.x;
		printf("\nasdasdan");

		for (int i = 0; i < N; i++)
		{
			for (int j = 0; j < N; j++)
			{
				for (int k = 0; k < N; k++)
				{
					C->rows[i].vec[j] += A->rows[k].vec[j] * B->rows[i].vec[k];
				}
			}
		}
		printf("\n--fgdfgdfgd--\n");
	}

	void CudaMath::MatMul44Test_cpu2()
	{
		for (int i = 0; i < 50; i++)
		{
			srand((unsigned int)time(NULL));


			Matrix44f A, B, C;
			ZERO_FLAT_MATRIX(C);
			Matrix44f transMat;
			float x = (float)rand() / (float)(RAND_MAX / 55.0f);
			float y = (float)rand() / (float)(RAND_MAX / 55.0f);
			float z = (float)rand() / (float)(RAND_MAX / 55.0f);



			float scale = (float)rand() / (float)(RAND_MAX / 55.0f);


			MAKE_SCALE(B, scale);

			glm::mat4 ref_scale = glm::scale(glm::mat4(1.0f), glm::vec3(scale, scale, scale));

			Vector3f transVec{ x, y, z };
			MAKE_TRANSLATION(A, transVec);


			glm::mat4 ref_trans = glm::translate(glm::mat4(1.0f), glm::vec3(x, y, z));
			//Matrix44f grid[4];




			/*MatMul44Batch_gpu CUDA_KERNEL(1, 1) (&A, &transMat, &C, 4);
			hipDeviceSynchronize();*/

			MatMul44_cpu(&A, &B, &C, 4);
			//hipDeviceSynchronize();

			glm::mat4 ref_result = glm::scale(ref_trans, glm::vec3(scale,scale,scale));
			
			for (int i = 0; i < 4; i++)
			{
				for (int j = 0; j < 4; j++)
				{

					printf("%f, ", C.rows[i].vec[j]);

				}
				printf("\n");
			}

			printf("\n----\n");
			for (int i = 0; i < 4; i++)\
			{
				for (int j = 0; j < 4; j++)
				{
					printf("%f, ", ref_result[i][j]);

				}
				printf("\n");
			}


			for (int i = 0; i < 4; i++)
				for (int j = 0; j < 4; j++)
				{
					if (C.rows[i].vec[j] != ref_result[i][j])
					{
						printf("after matrices dont match\n");
						return;
					}
				}
			printf("success\n");
		}
		
	}

	void MatMul44Test_cpu()
	{
		for (int i = 0; i < 50; i++)
		{
			srand((unsigned int)time(NULL));


			Matrix44f A, B, C;
			ZERO_FLAT_MATRIX(C);
			Matrix44f transMat;
			float x = (float)rand() / (float)(RAND_MAX / 55.0f);
			float y = (float)rand() / (float)(RAND_MAX / 55.0f);
			float z = (float)rand() / (float)(RAND_MAX / 55.0f);

			float scale = (float)rand() / (float)(RAND_MAX / 55.0f);
			MAKE_SCALE(B, scale);

			Vector3f transVec{ x, y, z };
			MAKE_TRANSLATION(A, transVec);

			Matrix44f* A_d, * B_d, * C_d;
			//hipMalloc((void**)&d_particles, common::ParticleData::num_particles * sizeof(common::particle_t));
			hipMalloc((void**)&A_d, sizeof(Matrix44f));
			hipMalloc((void**)&B_d, sizeof(Matrix44f));
			hipMalloc((void**)&C_d, sizeof(Matrix44f));

			hipMemcpy(A_d, (void*)&A, sizeof(Matrix44f), hipMemcpyHostToDevice);
			hipMemcpy(B_d, (void*)&B, sizeof(Matrix44f), hipMemcpyHostToDevice);
			hipMemcpy(C_d, (void*)&C, sizeof(Matrix44f), hipMemcpyHostToDevice);

			glm::mat4 ref_mats;
			//Matrix44f grid[4];


			for (int j = 0; j < 16; j++)
			{
				float x = (float)rand() / (float)(RAND_MAX / 55.0f);
				A.mat[j] = x;
			}

			for (int j = 0; j < 4; j++)
			{
				for (int k = 0; k < 4; k++)
					ref_mats[j][k] = A.rows[j].vec[k];
			}

			for (int i = 0; i < 4; i++)
				for (int j = 0; j < 4; j++)
				{
					if (A.rows[i].vec[j] != ref_mats[i][j])
					{
						printf("initial matrices dont match\n");
						return;
					}
				}
			printf("success\n");

			for (int i = 0; i < 4; i++)
			{
				for (int j = 0; j < 4; j++)
				{

					printf("%f, ", A.rows[i].vec[j]);

				}
				printf("\n");
			}

			printf("\n----\n");
			for (int i = 0; i < 4; i++)\
			{
				for (int j = 0; j < 4; j++)
				{
					printf("%f, ", ref_mats[i][j]);

				}
				printf("\n");
			}
			printf("\n----\n");
			printf("\n----\n");

			/*MatMul44Batch_gpu CUDA_KERNEL(1, 1) (&A, &transMat, &C, 4);
			hipDeviceSynchronize();*/
			hipError_t cudaerr = hipDeviceSynchronize();
			if (cudaerr != hipSuccess)
			{
				printf("1 kernel launch failed with error \"%s\".\n",
					hipGetErrorString(cudaerr));
				__debugbreak;
			}
			MatMul44_gpu CUDA_KERNEL(1, 1) (A_d, B_d, C_d, 4);
			//hipDeviceSynchronize();
			cudaerr = hipDeviceSynchronize();
			if (cudaerr != hipSuccess)
			{
				printf("2 kernel launch failed with error \"%s\".\n",
					hipGetErrorString(cudaerr));
				__debugbreak;
			}



			hipMemcpy((void*)&A, A_d, sizeof(Matrix44f), hipMemcpyDeviceToHost);
			hipMemcpy((void*)&B, B_d, sizeof(Matrix44f), hipMemcpyDeviceToHost);
			hipMemcpy((void*)&C, C_d, sizeof(Matrix44f), hipMemcpyDeviceToHost);




			glm::mat4 ref_result = glm::translate(glm::mat4(1.0f), glm::vec3(x, y, z)) * glm::scale(glm::mat4(1.0f), glm::vec3(scale));

			for (int i = 0; i < 4; i++)
			{
				for (int j = 0; j < 4; j++)
				{

					printf("%f, ", C.rows[i].vec[j]);

				}
				printf("\n");
			}

			printf("\n----\n");
			for (int i = 0; i < 4; i++)\
			{
				for (int j = 0; j < 4; j++)
				{
					printf("%f, ", ref_result[i][j]);

				}
				printf("\n");
			}


			for (int i = 0; i < 4; i++)
				for (int j = 0; j < 4; j++)
				{
					if (C.rows[i].vec[j] != ref_result[i][j])
					{
						printf("after matrices dont match\n");
						return;
					}
				}
			printf("success\n");

		}
		

	}

	






	
	
	

	void MatMulTest_cpu()
	{	
		printf("matmul cpu\n");

		
		int N = 4;

		Matrix44f A, B, C;

		ZERO_FLAT_MATRIX(C);
		//Vector3f svec{ 0.5f,0.5,0.5f };
		Vector3f tvec{ 2.0f,3.0f,4.0f };

		MAKE_TRANSLATION(A, tvec);
		//MAKE_SCALE(B, svec);
		MAKE_IDENTITY(B);

	

		dim3 threadsPerBlock(N, N);
		dim3 blocksPerGrid(1, 1);
		if (N * N > 16)
		{
			threadsPerBlock.x = 16;
			threadsPerBlock.y = 16;
			blocksPerGrid.x = ceil(double(N) / double(threadsPerBlock.x));
			blocksPerGrid.y = ceil(double(N) / double(threadsPerBlock.y));
		}

		//MatMul_gpu CUDA_KERNEL(blocksPerGrid, threadsPerBlock) (&A, &B, &C, N);

		hipDeviceSynchronize();


		glm::mat4 model = glm::translate(glm::mat4(1.0f), glm::vec3(2.0f, 3.0f, 4.0f));

		for (int i = 0; i < 4; i++)
		{
			for (int j = 0; j < 4; j++)
			{
				if (C.mat[j + i * 4] != model[i][j])
				{
					//std::cout << myMat.mat[j + i * 4] >> '\n';
					printf("error mult\n");
				}
			}

		}

	}

	
	

	__global__ void MakeIdentity_gpu_glm(Matrix44f* dest)
	{
		printf("makeId\n");
		glm::mat4 identity = glm::mat4(1.0f);
		memcpy(dest->mat, glm::value_ptr(identity), sizeof(float) * 16);
	}
	__global__ void MakeTranslation_gpu_glm(Matrix44f* dest, const glm::vec3& vec)
	{
		glm::mat4 source = glm::translate(glm::mat4(1.0f), vec);
		memcpy(dest->mat, glm::value_ptr(source), sizeof(float) * 16);
	}
	__global__ void MakeScale_gpu_glm(Matrix44f* dest, const glm::vec3& vec)
	{
		glm::mat4 source = glm::scale(glm::mat4(1.0f), vec);
		memcpy(dest->mat, glm::value_ptr(source), sizeof(float) * 16);
	}

	__global__ void TestIdentity_gpu()
	{
		printf("test identtityg\n");

		glm::mat4 baseCase = glm::mat4(1.0f);
		Matrix44f myMat;

		MAKE_IDENTITY(myMat);
		

		for (int i = 0; i < 4; i++)
		{
			for (int j = 0; j < 4; j++)
			{
				if (myMat.mat[j + i * 4] != baseCase[i][j])
				{
					//std::cout << myMat.mat[j + i * 4] >> '\n';
					printf("error identity\n");
				}
			}

		}

		//printf("success identity\n");

	}
	__global__ void TestTranslation_gpu()
	{
		printf("test trangs\n");

		glm::mat4 baseCase = glm::translate(glm::mat4(1.0f), glm::vec3(7.0f, 5.0f,9.0f));
		Matrix44f myMat;

		/*myMat.rows[0] = Vector4f({ 1.0f, 0.0f, 0.0f, 0.0f });
		myMat.rows[1] = Vector4f({ 0.0f, 1.0f, 0.0f, 0.0f });
		myMat.rows[2] = Vector4f({ 0.0f, 0.0f, 1.0f, 0.0f });
		myMat.rows[3] = Vector4f({ 2.0f, 3.0f, 4.0f, 1.0f });*/
		Vector3f tvec{ 7.0f,5.0f,9.0f };
		MAKE_TRANSLATION(myMat, tvec);

		for (int i = 0; i < 4; i++)
		{
			for (int j = 0; j < 4; j++)
			{
				if (myMat.mat[j + i * 4] != baseCase[i][j])
				{
					//std::cout << myMat.mat[j + i * 4] >> '\n';
					printf("error trans\n");
				}
			}
		}
	}
	__global__ void TestScale_gpu()
	{
		printf("test scaleg\n");

		glm::mat4 baseCase = glm::scale(glm::mat4(1.0f), glm::vec3(7.0f, 5.0f, 99.0f));
		Matrix44f myMat;
		Vector3f vec = Vector3f{ 7.0f,5.0f, 99.0f };
		/*vec.x = 1.0f;
		vec.y = 2.0f;
		vec.z = 3.0f;*/
		printf("x: {%f}, y: {%f}, z: {%f}\n\n", vec.vec[0], vec.vec[1], vec.vec[2]);

		//MAKE_SCALE(myMat, vec);

	/*	myMat.rows[0] = Vector4f({ 2.0f, 0.0f, 0.0f, 0.0f });
		myMat.rows[1] = Vector4f({ 0.0f, 3.0f, 0.0f, 0.0f });
		myMat.rows[2] = Vector4f({ 0.0f, 0.0f, 4.0f, 0.0f });
		myMat.rows[3] = Vector4f({ 0.0f, 0.0f, 0.0f, 1.0f });*/

		for (int i = 0; i < 4; i++)
		{
			for (int j = 0; j < 4; j++)
			{
				if (myMat.mat[j + i * 4] != baseCase[i][j])
				{
					printf("error scale\n");

				}
			}
		}
	}

	

	

	void MakeIdentity_cpu(Matrix44f* dest)
	{
		printf("makeId_cpu\n");
		hipDeviceSynchronize();

		//MakeIdentity_gpu CUDA_KERNEL(1, 1)(dest);
		//MakeIdentity_gpu (dest);
		hipDeviceSynchronize();

	}



	void MakeTranslation_cpu(Matrix44f* dest, const glm::vec3& vec)
	{
		printf("maketrans_cpu\n");
		//hipDeviceSynchronize();

		//MakeTranslation_gpu CUDA_KERNEL(1, 1)(dest, vec);
		//MakeTranslation_gpu (dest, vec);

		//hipDeviceSynchronize();
	}
	void MakeScale_cpu(Matrix44f* dest, const glm::vec3& vec)
	{
		printf("makescale_cpu\n");
		//hipDeviceSynchronize();

		//MakeScale_gpu (dest, vec);
		//MakeScale_gpu CUDA_KERNEL(1, 1)(dest, vec);
		//hipDeviceSynchronize();
	}

	void TestIdentity_cpu()
	{
		printf("test id cpu\n");
		hipDeviceSynchronize();

		TestIdentity_gpu CUDA_KERNEL(1, 1)();
		hipDeviceSynchronize();


	}
	void TestTranslation_cpu()
	{
		printf("test trans cpu\n");
		hipDeviceSynchronize();

		TestTranslation_gpu CUDA_KERNEL(1, 1)();
		hipDeviceSynchronize();
	}
	void TestScale_cpu()
	{
		printf("test scale cpu\n");
		hipDeviceSynchronize();

		TestScale_gpu CUDA_KERNEL(1, 1)();
		hipDeviceSynchronize();
	}
}